#include "hip/hip_runtime.h"
#include "Graph.h"
#include "Timer.h"
#include "QuickMapping.h"
#include <hip/hip_runtime.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

typedef struct {
    int *d_vertexOffset;
    int *d_adjacencyList;
    int *d_degree;
} GraphGPU;

typedef struct {
    int *h_vertexOffset;
    int *h_adjacencyList;
    int *h_degree;
} GraphCPU;

typedef struct {
    int *d_k;                   // TODO Migrate to __const__
    int *d_id;
    int *d_jobsPerWarp;         // TODO Migrate to __const__
    int *d_jobs;
    int *d_currentJob;
    int *d_currentPosOfJob;
    int *d_validJobs;
    int *d_numberOfExtensions;
    int *d_extensions;
    int *d_extensionSources;
    int *d_currentPos;
    int *d_warpSize;            
    int *d_extensionsOffset;
    int *d_extensionsLength;
    unsigned long* d_result;
    unsigned long long *d_hashPerWarp;
    long unsigned int *d_quickToCgLocal;
    int *d_numberOfCgs;
    long unsigned int *d_localSubgraphInduction;
} EnumerationGPU;

typedef struct {
    unsigned long* h_result;
    int *h_jobs;
    int *h_currentJob;
    int *h_currentPosOfJob;
    int *h_validJobs;
    int *h_extensionsOffset;
    unsigned long long *h_hashPerWarp;
    long unsigned int *h_hashGlobal;
} EnumerationCPU;


typedef struct {
    // Input
    int h_k;
    int h_numberOfActiveThreads;
    int h_blockSize;
    int h_numberOfSMs;
    int h_jobsPerWarp;

    // Others
    int h_warpSize;
    int h_numberOfBlocks;
    int h_warpsPerBlock;
    int h_numberOfWarps;
    
    int h_maxVertexId;
    int h_maxDegree;
    int h_numberOfEdges;
    int h_extensionsLength;
    int h_initialJobsPerWarp;
    int h_theoreticalJobsPerWarp;
} ConstantsCPU;


void initializeCpuDataStructures(Graph *graphReader, ConstantsCPU *constantsCPU, GraphCPU *graphCPU, EnumerationCPU *enumerationCPU, QuickMapping *quickMapping);
void initializeConstantsCpu(ConstantsCPU *constantsCPU, Graph *graphReader);
void releaseCpuDataStructures(GraphCPU *graphCPU, EnumerationCPU *enumerationCPU);
void initializeGpuDataStructures(ConstantsCPU *constantsCPU, GraphCPU* graphCPU, GraphGPU *graphGPU, EnumerationCPU *enumerationCPU, EnumerationGPU *enumerationGPU, QuickMapping *quickMapping);
void releaseGpuDataStructures(GraphGPU *graphGPU, EnumerationGPU *enumerationGPU);


typedef struct {
    int tid;
    int wid;
    int lane;
    int k;
    int offsetWarp;
    int offsetExtensions;
    int offsetInductions;
    int offsetHash;
} GPULocalVariables;


__device__ int roundToWarpSize(int value, int warpSize) {
    return ((int)ceilf((float)value / (float)warpSize)) * warpSize;
}

__device__ int neighbour(int vertexId, int relativePosition, GraphGPU *graph) {
    return graph->d_adjacencyList[graph->d_vertexOffset[vertexId]+relativePosition];
}

__device__ int getCurrentJob(GPULocalVariables *variables, EnumerationGPU *enumerationGPU) {
    return enumerationGPU->d_currentJob[variables->wid];
}

__device__ int getValidJobs(GPULocalVariables *variables, EnumerationGPU *enumerationGPU) {
    return enumerationGPU->d_validJobs[variables->wid];
}

__device__ int getCurrentPosOfJob(GPULocalVariables *variables, EnumerationGPU *enumerationGPU) {
    return enumerationGPU->d_currentPosOfJob[variables->wid*(*enumerationGPU->d_jobsPerWarp)+getCurrentJob(variables, enumerationGPU)];
}

__device__ int getJob(GPULocalVariables *variables, EnumerationGPU *enumerationGPU) {
    return enumerationGPU->d_jobs[variables->wid*(*enumerationGPU->d_jobsPerWarp)*(32) + getCurrentJob(variables, enumerationGPU)*(32) + variables->lane];
}


__device__ void popJob(GPULocalVariables *variables, EnumerationGPU *enumerationGPU) {
    if(variables->k != -1)
        return;
    
    if(getCurrentJob(variables, enumerationGPU) >= getValidJobs(variables, enumerationGPU)) {
        // No more jobs... warp is going to quit.
        // Update status and smid variables, when load-balancing comes up.
    }
    else {
        variables->k = getCurrentPosOfJob(variables, enumerationGPU);
        enumerationGPU->d_id[variables->offsetWarp+variables->lane] = getJob(variables, enumerationGPU);
        enumerationGPU->d_localSubgraphInduction[variables->offsetInductions+variables->lane] = 0;
        enumerationGPU->d_numberOfExtensions[variables->offsetWarp+variables->lane] = 0; // initializeNumberOfExtensions(0)
        enumerationGPU->d_numberOfExtensions[variables->offsetWarp+variables->k] = -1; // setNumberOfExtensions(-1)
        enumerationGPU->d_currentJob[variables->wid]++; // increaseJob()
    }
}

__device__ void backward(GPULocalVariables *variables, EnumerationGPU *enumerationGPU) {
    variables->k = variables->k - 1;
    if(variables->k < 0) {
        popJob(variables, enumerationGPU);
    } 
}

__device__ void forward(GPULocalVariables *variables, EnumerationGPU *enumerationGPU, GraphGPU *graph) {
    int numberOfExtensions = enumerationGPU->d_numberOfExtensions[variables->offsetWarp+variables->k];
    int localOffsetExtensions = variables->offsetExtensions + enumerationGPU->d_extensionsOffset[variables->k];
    int nextEmbeddingID = enumerationGPU->d_extensions[localOffsetExtensions+numberOfExtensions-1];
    enumerationGPU->d_numberOfExtensions[variables->offsetWarp+variables->k]--;
    variables->k = variables->k + 1;
    enumerationGPU->d_id[variables->offsetWarp+variables->k] = nextEmbeddingID;
    enumerationGPU->d_numberOfExtensions[variables->offsetWarp+variables->k] = -1;

    if(variables->k >= 2) {
        // (((dm_k()-2)*(2+dm_k()-1))/2) -> Offset created by previous inductions (sum of PA starting in 2)
        int localOffsetInduction = (((variables->k-2)*(2+variables->k-1))/2);

        unsigned long quickPattern = 0;
        for(int i = 0, currentPow = powf(2,localOffsetInduction), found = 0, currentDegree,  vertexId ; i < variables->k ; i++, currentPow*=2) {
            vertexId = enumerationGPU->d_id[variables->offsetWarp+i];
            currentDegree = graph->d_degree[vertexId];

            for(int warpPosition = variables->lane ; warpPosition < roundToWarpSize(currentDegree, 32) && !found ; warpPosition += 32)
                found = __any_sync(0xffffffff, warpPosition < currentDegree && neighbour(vertexId, warpPosition, graph) == nextEmbeddingID ? 1 : 0);
            
            quickPattern += (found*currentPow);
        }
        enumerationGPU->d_localSubgraphInduction[variables->offsetInductions+variables->k] = enumerationGPU->d_localSubgraphInduction[variables->offsetInductions+variables->k-1] + quickPattern;
    }
}

__global__ void motifs(GraphGPU *graph, EnumerationGPU *enumerationGPU) {
    GPULocalVariables variables;

    variables.tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    variables.wid = variables.tid / 32;
    variables.lane = threadIdx.x & 0x1f;
    variables.offsetWarp = variables.wid * 32;
    variables.offsetExtensions = variables.wid * *(enumerationGPU->d_extensionsLength);
    variables.offsetInductions = variables.wid * 32;
    variables.offsetHash = variables.wid * *(enumerationGPU->d_numberOfCgs);
    
    /*********************/
    /****  dm_start() ****/
    enumerationGPU->d_result[variables.wid] = 0;                                                    // result[dm_getWid()] = 0;
    variables.k = -1;                                                                               // dm_k(currentPos[dm_getWid()]);
    popJob(&variables, enumerationGPU);                                                             // dm_popJob();
    /*********************/

    while(variables.k >= 0) {                                                                       // while(dm_active() && dm_gpuIsBalanced())
        if(enumerationGPU->d_numberOfExtensions[variables.offsetWarp+variables.k] == -1) {          // if(dm_numberOfExtensions() == -1)
            /**************************************************/
            /** [BEGIN] generateUniqueExtensions(0, dm_k()) **/
            int localOffsetExtensions = variables.offsetExtensions + enumerationGPU->d_extensionsOffset[variables.k];
            int currentOffsetExtensions = 0;
            int v0 = enumerationGPU->d_id[variables.offsetWarp];
            unsigned long pos, actives, totalActives, activesOnMyRight, idlesOnMyRight;

            enumerationGPU->d_numberOfExtensions[variables.offsetWarp+variables.k] = 0;

            for(int i = 0, currentVertexDegree, currentVertexDegreeRounded ; i <= variables.k ; i++) {
                int id = enumerationGPU->d_id[variables.offsetWarp+i];
                currentVertexDegree = graph->d_degree[id];
                currentVertexDegreeRounded = roundToWarpSize(currentVertexDegree, 32);

                for(int warpPosition = variables.lane, currentNeighbour ; warpPosition < currentVertexDegreeRounded ;  warpPosition += 32) {
                    currentNeighbour = neighbour(id, warpPosition, graph);
                    currentNeighbour = warpPosition < currentVertexDegree && currentNeighbour > v0 ? currentNeighbour : -1;
                    bool found = false;
                    for(int warpPosition = 0 ; warpPosition < currentOffsetExtensions && !found && currentNeighbour != -1 ; warpPosition++)
                        found = enumerationGPU->d_extensions[localOffsetExtensions+warpPosition] == currentNeighbour;
                    __syncwarp();
                    currentNeighbour = found ? -1 : currentNeighbour;
                    
                    actives = __ballot_sync(0xffffffff, currentNeighbour == -1 ? 0 : 1);
                    totalActives = __popc(actives);
                    actives = (actives << ((unsigned int)32-(unsigned int)variables.lane));
                    activesOnMyRight = __popc(actives);
                    idlesOnMyRight = variables.lane - activesOnMyRight;

                    pos = currentNeighbour != -1 ? activesOnMyRight : totalActives + idlesOnMyRight;
                    enumerationGPU->d_extensions[localOffsetExtensions+currentOffsetExtensions+pos] = currentNeighbour;
                    enumerationGPU->d_extensionSources[localOffsetExtensions+currentOffsetExtensions+pos] = i;
                    currentOffsetExtensions += totalActives;
                }
            }
            enumerationGPU->d_numberOfExtensions[variables.offsetWarp+variables.k] = currentOffsetExtensions;
            /** [END] generateUniqueExtensions(0, dm_k())   **/

            /**************************************************/
            /********* [BEGIN] canonicalFilter() ***************/
            if(variables.k > 0) {
                int currentOffsetExtensionsNew;
                for(int i = 1, target ; i <= variables.k ; i++) {
                    target = enumerationGPU->d_id[variables.offsetWarp+i];;
                    currentOffsetExtensionsNew = 0;
                    for(int warpPosition = variables.lane, ext, src ; warpPosition < roundToWarpSize(currentOffsetExtensions, 32) ; warpPosition += 32) {
                        ext = warpPosition < currentOffsetExtensions ? enumerationGPU->d_extensions[localOffsetExtensions+warpPosition] : -1;
                        src = warpPosition < currentOffsetExtensions ? enumerationGPU->d_extensionSources[localOffsetExtensions+warpPosition] : -1;
                        
                        ext = (i > src && ext <= target) || (i < src && ext == target) ? -1 : ext; 

                        actives = __ballot_sync(0xffffffff, ext == -1 ? 0 : 1);
                        totalActives = __popc(actives);
                        actives = (actives << ((unsigned int)32-(unsigned int)variables.lane));
                        activesOnMyRight = __popc(actives);
                        idlesOnMyRight = variables.lane - activesOnMyRight;

                        pos = ext != -1 ? activesOnMyRight : totalActives + idlesOnMyRight;
                        enumerationGPU->d_extensions[localOffsetExtensions+currentOffsetExtensionsNew+pos] = ext;
                        enumerationGPU->d_extensionSources[localOffsetExtensions+currentOffsetExtensionsNew+pos] = src;
                        currentOffsetExtensionsNew += totalActives;
                    }
                    currentOffsetExtensions = currentOffsetExtensionsNew;
                }
                enumerationGPU->d_numberOfExtensions[variables.offsetWarp+variables.k] = currentOffsetExtensions;
            }
            /********** [END] canonicalFilter() ***************/
        }

        int numberOfExtensions = enumerationGPU->d_numberOfExtensions[variables.offsetWarp+variables.k];
        if(numberOfExtensions != 0) {
            if(variables.k == *enumerationGPU->d_k-2) {
                /************************************/
                /* [BEGIN] accumulateValidSubgraphs */
                enumerationGPU->d_result[variables.wid] += numberOfExtensions;
                /************************************/

                /*********************************/ 
                /*[BEGIN] aggregateValidSubgraphs*/
                int localOffsetExtensions = variables.offsetExtensions + enumerationGPU->d_extensionsOffset[variables.k];
                unsigned long quickPattern = 0;
                int localOffsetInduction = (((variables.k-1)*(2+variables.k))/2);

                for(int warpPosition = variables.lane, nextEmbeddingId ; warpPosition < roundToWarpSize(numberOfExtensions, 32) ; warpPosition+=32) {
                    if(warpPosition < numberOfExtensions) {
                        nextEmbeddingId = enumerationGPU->d_extensions[localOffsetExtensions+warpPosition];

                        // Final induction (quick pattern)
                        quickPattern = 0;
                        for(int j = 0, currentPow = powf(2,localOffsetInduction), found, currentDegree, vertexId ; j <= variables.k ; j++, currentPow*=2) {
                            found = 0; 
                            vertexId = enumerationGPU->d_id[variables.offsetWarp+j];
                            currentDegree = graph->d_degree[vertexId];
                            for(int warpPosition = 0 ; warpPosition < currentDegree && !found ; warpPosition++) {
                                if(neighbour(vertexId, warpPosition, graph) == nextEmbeddingId)
                                    found = 1;
                            }
                            quickPattern += (found*currentPow);
                        }
                    }
                    __syncwarp();
                    if(warpPosition < numberOfExtensions) {
                        quickPattern += enumerationGPU->d_localSubgraphInduction[variables.offsetInductions+variables.k];
                        atomicAdd_block(&(enumerationGPU->d_hashPerWarp[variables.offsetHash+enumerationGPU->d_quickToCgLocal[quickPattern]]), 1);
                    }
                    __syncwarp();
                }
                /*[END] aggregateValidSubgraphs*/

                backward(&variables, enumerationGPU);
            }
            else {
                forward(&variables, enumerationGPU, graph);
            }
        }
        else {
            backward(&variables, enumerationGPU);
        }
    }

    // Silly condition just to check the kernel finished without errors 
    if(variables.tid == 0)
        printf("After all, we are only human...\n");
}

int main(int argc, const char** argv) {
    printf("Usage: ./motifs_HAND_WC graphFile k threads blockSize\n");
    printf("\t graphFile: \t url of graph dataset\n");
    printf("\t k: \t\t clique size\n");
    printf("\t threads: \t amount of GPU threads (recommended: 102400)\n");
    printf("\t blockSize: \t amount of threads per block (recommended: 256)\n");
    if(argc != 5) {
        printf("\nWrong amount of parameters!\n");
        printf("Exiting...\n");
        exit(1);
    }

    ConstantsCPU *constantsCPU = (ConstantsCPU*)malloc(sizeof(ConstantsCPU));
    Timer timer;

    Graph *graphReader = new Graph(argv[1]);
    constantsCPU->h_k = atoi(argv[2]);
    constantsCPU->h_numberOfActiveThreads = atoi(argv[3]);
    constantsCPU->h_blockSize = atoi(argv[4]);
    constantsCPU->h_numberOfSMs = 80;
    constantsCPU->h_jobsPerWarp = 16;    
    initializeConstantsCpu(constantsCPU, graphReader);

    QuickMapping *quickMapping = new QuickMapping(constantsCPU->h_k);
    
    GraphCPU *graphCPU = (GraphCPU*)malloc(sizeof(GraphCPU));
    EnumerationCPU *enumerationCPU = (EnumerationCPU*)malloc(sizeof(EnumerationCPU));
    initializeCpuDataStructures(graphReader, constantsCPU, graphCPU, enumerationCPU, quickMapping);

    GraphGPU *graphGPU = (GraphGPU*)malloc(sizeof(GraphGPU));
    EnumerationGPU *enumerationGPU = (EnumerationGPU*)malloc(sizeof(EnumerationGPU));
    initializeGpuDataStructures(constantsCPU, graphCPU, graphGPU, enumerationCPU, enumerationGPU, quickMapping);

    // printf("Number of blocks: %d, block size: %d\n", constantsCPU->h_numberOfBlocks, constantsCPU->h_blockSize);

    GraphGPU *d_graphGPU;
    gpuErrchk(hipMalloc((void**)&d_graphGPU, sizeof(GraphGPU)));
    gpuErrchk(hipMemcpy(d_graphGPU, graphGPU, sizeof(GraphGPU), hipMemcpyHostToDevice));

    EnumerationGPU *d_enumerationGPU;
    gpuErrchk(hipMalloc((void**)&d_enumerationGPU, sizeof(EnumerationGPU)));
    gpuErrchk(hipMemcpy(d_enumerationGPU, enumerationGPU, sizeof(EnumerationGPU), hipMemcpyHostToDevice));

    timer.play();
    motifs<<<constantsCPU->h_numberOfBlocks, constantsCPU->h_blockSize>>>(d_graphGPU, d_enumerationGPU);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    
    /***********************/
    /* Organizing results. */
    gpuErrchk(hipMemcpy(enumerationCPU->h_result, enumerationGPU->d_result, constantsCPU->h_numberOfWarps*sizeof(unsigned long), hipMemcpyDeviceToHost));
    unsigned long result = 0;
    for(int i = 0 ; i < constantsCPU->h_numberOfWarps ; i++) {
        result += enumerationCPU->h_result[i];
    }
    gpuErrchk(hipMemcpy(enumerationCPU->h_hashPerWarp, enumerationGPU->d_hashPerWarp, constantsCPU->h_numberOfWarps * quickMapping->numberOfCgs * sizeof(unsigned long long), hipMemcpyDeviceToHost));
    for(int i = 0 ; i < constantsCPU->h_numberOfWarps ; i++) {
        for(int j = 0 ; j < quickMapping->numberOfCgs ; j++) {
            enumerationCPU->h_hashGlobal[j] += enumerationCPU->h_hashPerWarp[i*quickMapping->numberOfCgs+j];
        }
    }
    long unsigned int validSubgraphs = 0;
    for(int i = 0 ; i < quickMapping->numberOfCgs ; i++) {
        if(enumerationCPU->h_hashGlobal[i] > 0) {
            validSubgraphs += enumerationCPU->h_hashGlobal[i];
        }
    }
    /************************/
    timer.pause();

    printf("Result: %lu (Aggregation: %s), tempo: %f\n", result, result == validSubgraphs ? "MATCH" : "MISMATCH", timer.getElapsedTimeInSeconds());

    /*************************************************/
    /************** Memory release *******************/
    releaseGpuDataStructures(graphGPU, enumerationGPU);
    releaseCpuDataStructures(graphCPU, enumerationCPU);
    delete graphReader;
    delete quickMapping;
    free(constantsCPU);
    /*************************************************/


    return 0;
}

void initializeGpuDataStructures(ConstantsCPU *constantsCPU, GraphCPU *graphCPU, GraphGPU *graphGPU, EnumerationCPU *enumerationCPU, EnumerationGPU *enumerationGPU, QuickMapping *quickMapping) {
    /***************************************/
    /************ Graph related ************/
    gpuErrchk(hipMalloc((void**)&graphGPU->d_vertexOffset, (constantsCPU->h_maxVertexId+2)*sizeof(int)));
    gpuErrchk(hipMalloc((void**)&graphGPU->d_adjacencyList, (constantsCPU->h_numberOfEdges*2 + (constantsCPU->h_maxVertexId+1)) * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&graphGPU->d_degree, (constantsCPU->h_maxVertexId+1)*sizeof(int)));

    gpuErrchk(hipMemcpy(graphGPU->d_vertexOffset, graphCPU->h_vertexOffset, (constantsCPU->h_maxVertexId+2)*sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(graphGPU->d_adjacencyList, graphCPU->h_adjacencyList, (constantsCPU->h_numberOfEdges*2 + (constantsCPU->h_maxVertexId+1)) * sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(graphGPU->d_degree, graphCPU->h_degree, (constantsCPU->h_maxVertexId+1)*sizeof(int), hipMemcpyHostToDevice));
    /***************************************/

    /***************************************/
    /******** Enumeration related **********/
    gpuErrchk(hipMalloc((void**)&(enumerationGPU->d_k), sizeof(int)));
    gpuErrchk(hipMalloc((void**)&(enumerationGPU->d_extensionsLength), sizeof(int)));
    gpuErrchk(hipMalloc((void**)&(enumerationGPU->d_id), constantsCPU->h_numberOfWarps * constantsCPU->h_warpSize * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&(enumerationGPU->d_jobsPerWarp), sizeof(int)));
    gpuErrchk(hipMalloc((void**)&(enumerationGPU->d_jobs), constantsCPU->h_numberOfWarps * constantsCPU->h_theoreticalJobsPerWarp * constantsCPU->h_warpSize * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&(enumerationGPU->d_currentJob), constantsCPU->h_numberOfWarps * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&(enumerationGPU->d_currentPosOfJob), constantsCPU->h_numberOfWarps * constantsCPU->h_theoreticalJobsPerWarp * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&(enumerationGPU->d_validJobs), constantsCPU->h_numberOfWarps * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&(enumerationGPU->d_numberOfExtensions), constantsCPU->h_numberOfWarps * constantsCPU->h_warpSize * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&(enumerationGPU->d_extensions), constantsCPU->h_numberOfWarps * constantsCPU->h_extensionsLength * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&(enumerationGPU->d_extensionSources), constantsCPU->h_numberOfWarps * constantsCPU->h_extensionsLength * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&(enumerationGPU->d_currentPos), constantsCPU->h_numberOfWarps * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&(enumerationGPU->d_result), constantsCPU->h_numberOfWarps * sizeof(unsigned long)));
    gpuErrchk(hipMalloc((void**)&(enumerationGPU->d_warpSize), sizeof(int)));
    gpuErrchk(hipMalloc((void**)&(enumerationGPU->d_extensionsOffset), (constantsCPU->h_k-1) * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&(enumerationGPU->d_hashPerWarp), constantsCPU->h_numberOfWarps * quickMapping->numberOfCgs * sizeof(unsigned long long)));
    gpuErrchk(hipMalloc((void**)&(enumerationGPU->d_quickToCgLocal), quickMapping->numberOfQuicks * sizeof(long unsigned int)));
    gpuErrchk(hipMalloc((void**)&(enumerationGPU->d_numberOfCgs), sizeof(int)));
    gpuErrchk(hipMalloc((void**)&(enumerationGPU->d_localSubgraphInduction), constantsCPU->h_numberOfWarps * constantsCPU->h_warpSize * sizeof(long unsigned int)));

    gpuErrchk(hipMemcpy(enumerationGPU->d_jobsPerWarp, &(constantsCPU->h_theoreticalJobsPerWarp), sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(enumerationGPU->d_jobs, enumerationCPU->h_jobs, constantsCPU->h_numberOfWarps * constantsCPU->h_theoreticalJobsPerWarp * constantsCPU->h_warpSize * sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(enumerationGPU->d_currentJob, enumerationCPU->h_currentJob, constantsCPU->h_numberOfWarps * sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(enumerationGPU->d_validJobs, enumerationCPU->h_validJobs, constantsCPU->h_numberOfWarps * sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(enumerationGPU->d_currentPosOfJob, enumerationCPU->h_currentPosOfJob, constantsCPU->h_numberOfWarps * constantsCPU->h_theoreticalJobsPerWarp * sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(enumerationGPU->d_warpSize, &(constantsCPU->h_warpSize), sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(enumerationGPU->d_extensionsOffset, enumerationCPU->h_extensionsOffset, (constantsCPU->h_k-1) * sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(enumerationGPU->d_k, &constantsCPU->h_k, sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(enumerationGPU->d_extensionsLength, &constantsCPU->h_extensionsLength, sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(enumerationGPU->d_quickToCgLocal, quickMapping->quickToCgLocal, quickMapping->numberOfQuicks * sizeof(long unsigned int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(enumerationGPU->d_numberOfCgs, &(quickMapping->numberOfCgs), sizeof(int), hipMemcpyHostToDevice));

    gpuErrchk(hipMemset(enumerationGPU->d_hashPerWarp, 0, constantsCPU->h_numberOfWarps * quickMapping->numberOfCgs * sizeof(unsigned long long)));
    gpuErrchk(hipMemset(enumerationGPU->d_localSubgraphInduction, 0, constantsCPU->h_numberOfWarps * constantsCPU->h_warpSize * sizeof(long unsigned int)));
    /***************************************/
}

void releaseGpuDataStructures(GraphGPU *graphGPU, EnumerationGPU *enumerationGPU) {
    // Graph related
    hipFree(graphGPU->d_vertexOffset);
    hipFree(graphGPU->d_adjacencyList);
    hipFree(graphGPU->d_degree); 
    // free(graphGPU);

    // Enumeration related
    hipFree(enumerationGPU->d_k);
    hipFree(enumerationGPU->d_extensionsLength);
    hipFree(enumerationGPU->d_id);
    hipFree(enumerationGPU->d_jobsPerWarp);
    hipFree(enumerationGPU->d_jobs);
    hipFree(enumerationGPU->d_currentJob);
    hipFree(enumerationGPU->d_currentPosOfJob); 
    hipFree(enumerationGPU->d_validJobs);
    hipFree(enumerationGPU->d_numberOfExtensions);
    hipFree(enumerationGPU->d_extensions); 
    hipFree(enumerationGPU->d_currentPos); 
    hipFree(enumerationGPU->d_result);
    hipFree(enumerationGPU->d_warpSize);
    hipFree(enumerationGPU->d_extensionsOffset);
    hipFree(enumerationGPU->d_hashPerWarp);
    hipFree(enumerationGPU->d_quickToCgLocal);
    hipFree(enumerationGPU->d_numberOfCgs);
    hipFree(enumerationGPU->d_extensionSources);
    hipFree(enumerationGPU->d_localSubgraphInduction);
    // free(enumerationGPU);
}

void initializeCpuDataStructures(Graph *graphReader, ConstantsCPU *constantsCPU, GraphCPU *graphCPU, EnumerationCPU *enumerationCPU, QuickMapping *quickMapping) {
    graphCPU->h_vertexOffset = (int*)malloc((constantsCPU->h_maxVertexId) * sizeof(int));
    graphCPU->h_adjacencyList = (int*)malloc(((constantsCPU->h_numberOfEdges*2) + (constantsCPU->h_maxVertexId+1)) * sizeof(int));
    graphCPU->h_degree = (int*)malloc((constantsCPU->h_maxVertexId+1)*sizeof(int));

    // Initialize CSR graph data structures for GPU
    int offset = 0;
    for(int vertexId = 0 ; vertexId <= graphReader->getMaxVertexId() ; vertexId++) {
        graphCPU->h_vertexOffset[vertexId] = offset;
        // printf("%d, vertexOffset: %d\n", vertexId, h_vertexOffset[vertexId]);
        for(std::set<int>::iterator itEdges = graphReader->getNeighbours(vertexId)->begin() ; itEdges != graphReader->getNeighbours(vertexId)->end() ; itEdges++)
            graphCPU->h_adjacencyList[offset++] = *itEdges;
        graphCPU->h_adjacencyList[offset++] = -1;

        graphCPU->h_degree[vertexId] = graphReader->getNeighbours(vertexId)->size();
        // printf("%d, vertexOffset: %d, degree: %d\n", vertexId, h_vertexOffset[vertexId], h_degree[vertexId]);
    }
    graphCPU->h_vertexOffset[graphReader->getMaxVertexId()+1] = graphCPU->h_vertexOffset[graphReader->getMaxVertexId()]+graphCPU->h_degree[graphReader->getMaxVertexId()]+1;

    enumerationCPU->h_hashPerWarp = (unsigned long long*)malloc(constantsCPU->h_numberOfWarps*quickMapping->numberOfCgs * sizeof(unsigned long long));
    enumerationCPU->h_hashGlobal = (long unsigned int*)malloc(quickMapping->numberOfCgs * sizeof(long unsigned int));
    enumerationCPU->h_result = (unsigned long*)malloc(constantsCPU->h_numberOfWarps * sizeof(unsigned long));
    enumerationCPU->h_currentJob = (int*)malloc(constantsCPU->h_numberOfWarps * sizeof(int));
    enumerationCPU->h_currentPosOfJob = (int*)malloc(constantsCPU->h_numberOfWarps * constantsCPU->h_theoreticalJobsPerWarp * sizeof(int));
    enumerationCPU->h_validJobs = (int*)malloc(constantsCPU->h_numberOfWarps * sizeof(int));
    enumerationCPU->h_jobs = (int*)malloc(constantsCPU->h_numberOfWarps * constantsCPU->h_theoreticalJobsPerWarp * constantsCPU->h_warpSize * sizeof(int));

    for(int i = 0 ; i < constantsCPU->h_numberOfWarps ; i++) {
        enumerationCPU->h_currentJob[i] = 0;
        enumerationCPU->h_validJobs[i] = 0;
        for(int j = 0 ; j < quickMapping->numberOfCgs ; j++)
            enumerationCPU->h_hashPerWarp[i*quickMapping->numberOfCgs + j] = 0;
    }
    for(int i = 0 ; i < quickMapping->numberOfCgs ; i++)
        enumerationCPU->h_hashGlobal[i] = 0;

    for(int round = 0 ; round < constantsCPU->h_initialJobsPerWarp ; round++) {
        for(int i = 0 ; i < constantsCPU->h_numberOfWarps ; i++) {
            int jobId = round*constantsCPU->h_numberOfWarps+i;
            if(jobId <= constantsCPU->h_maxVertexId) {
                enumerationCPU->h_validJobs[i]++;
                enumerationCPU->h_jobs[i*constantsCPU->h_theoreticalJobsPerWarp*constantsCPU->h_warpSize + round*constantsCPU->h_warpSize + 0] = jobId;
                enumerationCPU->h_currentPosOfJob[i*constantsCPU->h_theoreticalJobsPerWarp+round] = 0;
            } 
        }
    }

    constantsCPU->h_extensionsLength = 0;
    enumerationCPU->h_extensionsOffset = (int*)malloc(constantsCPU->h_k * sizeof(int));
    enumerationCPU->h_extensionsOffset[0] = 0;
    for(int k = 1, length ; k <= constantsCPU->h_k - 1 ; k++) {
        length = (int)ceilf(pow(2, ceilf(log2((float)(k * constantsCPU->h_maxDegree)))));
        constantsCPU->h_extensionsLength += length;
        if(k < constantsCPU->h_k - 1)
            enumerationCPU->h_extensionsOffset[k] = constantsCPU->h_extensionsLength;
    }
}

void releaseCpuDataStructures(GraphCPU *graphCPU, EnumerationCPU *enumerationCPU) {
    // free(graphCPU->h_vertexOffset);
    // free(graphCPU->h_adjacencyList);
    // free(graphCPU->h_degree);
    // free(graphCPU);

    // free(enumerationCPU->h_result);
    // free(enumerationCPU->h_currentJob);
    // free(enumerationCPU->h_currentPosOfJob);
    // free(enumerationCPU->h_validJobs);
    // free(enumerationCPU->h_jobs);
    // free(enumerationCPU->h_extensionsOffset);
    // free(enumerationCPU->h_hashPerWarp)
    // free(enumerationCPU->h_hashGlobal);
    // free(enumerationCPU);
}

void initializeConstantsCpu(ConstantsCPU* constantsCPU, Graph* graphReader) {
    constantsCPU->h_maxVertexId = graphReader->getMaxVertexId();
    constantsCPU->h_numberOfEdges = graphReader->getNumberOfEdges();
    constantsCPU->h_maxDegree = graphReader->getMaxDegree();

    constantsCPU->h_warpSize = 32;
    constantsCPU->h_numberOfBlocks = ceil(constantsCPU->h_numberOfActiveThreads/(float)constantsCPU->h_blockSize);
    constantsCPU->h_warpsPerBlock = constantsCPU->h_blockSize / constantsCPU->h_warpSize;
    constantsCPU->h_numberOfWarps = constantsCPU->h_numberOfBlocks * constantsCPU->h_warpsPerBlock;

    constantsCPU->h_initialJobsPerWarp = ceil((constantsCPU->h_maxVertexId+1)/(float)constantsCPU->h_numberOfWarps);    
    constantsCPU->h_theoreticalJobsPerWarp = std::max(constantsCPU->h_initialJobsPerWarp, constantsCPU->h_jobsPerWarp);
}
