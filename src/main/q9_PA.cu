#include "hip/hip_runtime.h"
#include "Graph.h"
#include "Timer.h"
#include "Structs.cu"
#include "DuMatoCPU.h"
#include "DuMatoGPU.cu"
#include "Report.h"
#include <hip/hip_runtime.h>

__global__ void q9_PA(DataGPU *dataGPU) {
    DuMatoGPU DM_GPU;
    DM_GPU.start_q9(dataGPU);

    while(DM_GPU.active() && DM_GPU.balanced()) {
        if(DM_GPU.getCurrentNumberOfExtensions() == -1) {
            DM_GPU.extend_q9();
        }
        if(DM_GPU.getCurrentNumberOfExtensions() != 0) {
            if(DM_GPU.last_level()) {
                DM_GPU.aggregate_counter();
                DM_GPU.backward_q9();
            }
            else {
                DM_GPU.forward();
            }
        }
        else {
            DM_GPU.backward_q9();
        }
    }

    DM_GPU.end();
}

int main(int argc, const char** argv) {
    printf("Usage: %s graphFile k threads blockSize donationsPerWarp threshold\n", argv[0]);
    printf("\t graphFile: \t\t url of graph dataset\n");
    printf("\t k: \t\t\t clique size\n");
    printf("\t threads: \t\t amount of GPU threads (recommended: 102400)\n");
    printf("\t blockSize: \t\t amount of threads per block (recommended: 256)\n");
    printf("\t donationsPerWarp: \t amount of donations during load-balancing (recommended: 16)\n");
    printf("\t threshold: \t\t load-balancing threshold (recommended: 30)\n");

    if(argc != 7) {
        printf("\nWrong amount of parameters!\n");
        printf("Exiting...\n");
        exit(1);
    }

    Timer timerTOTAL;
    timerTOTAL.play("timerTOTAL");

    /*************************************************/
    /***************     Input    ********************/
    const char *datasetName = argv[1];
    int k = atoi(argv[2]);
    int numberOfActiveThreads = atoi(argv[3]);
    int blockSize = atoi(argv[4]);
    int numberOfSMs = 80;
    int jobsPerWarp = atoi(argv[5]);
    int globalThreshold = atoi(argv[6]);
    bool relabeling = false;
    bool patternAware = true;
    int rep = 0;
    /*************************************************/

    Timer timerIO;
    double timeLB = 0;

    timerIO.play("IO");
    DuMatoCPU *DM_CPU = new DuMatoCPU(datasetName, k, numberOfActiveThreads, blockSize, numberOfSMs, jobsPerWarp, q9_PA, globalThreshold, relabeling, patternAware);
    Report* report;
    if(rep)
        report = new Report(DM_CPU, 100);
    hipDeviceSynchronize();
    timerIO.pause("IO");

    Timer timerGPU;
    if(globalThreshold >= 100) {
        timerGPU.play("Kernel");
        DM_CPU->runKernel();
        if(rep)
            report->start();
        DM_CPU->waitKernel();
        if(rep)
            report->stop();
        timerGPU.pause("Kernel");
    }
    else {
        timerGPU.play("Kernel");
        DM_CPU->runKernel();
        if(rep)
            report->start();
        while(true) {
            if(DM_CPU->gpuIsIdle()) {
                printf("[gpuIsIdle] %.2f.\n", DM_CPU->dataCPU->h_percentageWarpsIdle);
                DM_CPU->stopKernel();
                Timer timerLB;
                timerLB.play();
                if(DM_CPU->rebalance()) {
                    timerLB.pause();
                    timeLB += timerLB.getElapsedTimeInSeconds();
                    DM_CPU->runKernel();
                }
                else
                    break;
            }
            DM_CPU->sleepFor(100);
        } 
        DM_CPU->waitKernel();
        if(rep)
            report->stop();
        timerGPU.pause("Kernel");
    }

    DM_CPU->outputAggregateCounter();
    timerTOTAL.pause("timerTOTAL");

    printf("Time (GPU): %f secs.\n", timerGPU.getElapsedTimeInSeconds());
    printf("Time (LB): %f secs,%f%%\n", timeLB, (timeLB/timerGPU.getElapsedTimeInSeconds())*100);
    printf("Time (IO): %f secs\n", timerIO.getElapsedTimeInSeconds());
    printf("Time (TOTAL): %f secs\n", timerTOTAL.getElapsedTimeInSeconds());
    // /*************************************************/
    /************** Memory release *******************/
    delete DM_CPU;
    if(rep)
        delete report;
    /*************************************************/

    return 0;
}
