#include "hip/hip_runtime.h"
#include "Graph.h"
#include "Timer.h"
#include "Structs.cu"
#include "DuMatoCPU.h"
#include "DuMatoGPU.cu"
#include "Report.h"
#include <hip/hip_runtime.h>

__global__ void motifs(DataGPU *dataGPU) {
    DuMatoGPU DM_GPU;
    DM_GPU.start_induce(dataGPU);

    while(DM_GPU.active() && DM_GPU.balanced()) {
        if(DM_GPU.getCurrentNumberOfExtensions() == -1) {
            DM_GPU.extend(0, DM_GPU.k());
            if(DM_GPU.k() > 0) {
                DM_GPU.canonicalFilter2();                
            }
        }
        if(DM_GPU.getCurrentNumberOfExtensions() != 0) {
            if(DM_GPU.last_level()) {
                DM_GPU.aggregate_pattern();
                DM_GPU.backward_induce();
            }
            else {
                DM_GPU.forward_induce();
            }
        }
        else {
            DM_GPU.backward_induce();
        }
    }

    DM_GPU.end();
}

int main(int argc, const char** argv) {
    printf("Usage: %s graphFile k threads blockSize donationsPerWarp threshold\n", argv[0]);
    printf("\t graphFile: \t\t url of graph dataset\n");
    printf("\t k: \t\t\t clique size\n");
    printf("\t threads: \t\t amount of GPU threads (recommended: 102400)\n");
    printf("\t blockSize: \t\t amount of threads per block (recommended: 256)\n");
    printf("\t donationsPerWarp: \t amount of donations during load-balancing (recommended: 16)\n");
    printf("\t threshold: \t\t load-balancing threshold (recommended: 30)\n");

    if(argc != 7) {
        printf("\nWrong amount of parameters!\n");
        printf("Exiting...\n");
        exit(1);
    }

    Timer timerTOTAL;
    timerTOTAL.play();

    /*************************************************/
    /***************     Input    ********************/
    const char *datasetName = argv[1];
    int k = atoi(argv[2]);
    int numberOfActiveThreads = atoi(argv[3]);
    int blockSize = atoi(argv[4]);
    int numberOfSMs = 80;
    int jobsPerWarp = atoi(argv[5]);
    int globalThreshold = atoi(argv[6]);
    int validateResults = 0;
    bool relabeling = true;
    int rep = 0;
    /*************************************************/

    Timer timerIO;
    Timer timerLB;

    timerIO.play();
    DuMatoCPU *DM_CPU = new DuMatoCPU(datasetName, k, numberOfActiveThreads, blockSize, numberOfSMs, jobsPerWarp, motifs, globalThreshold, relabeling);
    hipDeviceSynchronize();
    timerIO.pause();

    Report* report;
    if(rep)
        report = new Report(DM_CPU, 100);
    
    Timer timerGPU;

    if(globalThreshold >= 100) {
        timerGPU.play("Kernel");
        DM_CPU->runKernel();
        if(rep)
            report->start();
        DM_CPU->waitKernel();
        timerGPU.pause("Kernel");
        if(rep)
            report->stop();
    }
    else {
        timerGPU.play("Kernel");
        DM_CPU->runKernel();
        if(rep)
            report->start();
        while(true) {
            if(DM_CPU->gpuIsIdle()) {
                printf("[gpuIsIdle] %.2f.\n", DM_CPU->dataCPU->h_percentageWarpsIdle);
                timerLB.play();
                DM_CPU->stopKernel();
                if(DM_CPU->rebalance()) {
                    timerLB.pause();
                    DM_CPU->runKernel();
                }
                else {
                    timerLB.pause();
                    break;
                }
            }
            DM_CPU->sleepFor(100);
        } 
        DM_CPU->waitKernel();
        if(rep)
            report->stop();
        timerGPU.pause("Kernel");
    }

    DM_CPU->outputAggregatePattern();
    timerTOTAL.pause();

    if(validateResults) { 
        DM_CPU->validateAggregatePattern();
    }
    printf("Time (GPU): %f secs.\n", timerGPU.getElapsedTimeInSeconds());
    printf("Time (LB): %f secs.\n", timerLB.getElapsedTimeInSeconds());
    printf("Time (IO): %f secs\n", timerIO.getElapsedTimeInSeconds());
    printf("Time (TOTAL): %f secs\n", timerTOTAL.getElapsedTimeInSeconds());
    /*************************************************/
    /************** Memory release *******************/
    delete DM_CPU;
    if(rep)
        delete report;
    /*************************************************/

    return 0;
}
